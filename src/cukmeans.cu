#include "hip/hip_runtime.h"
#include "cukmeans.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
cukmeans::cukmeans()
{
    _feat_dim   = 0;
    _class_dim  = 0;
    _sample_num = 0;
}

cukmeans::~cukmeans()
{
}

int cukmeans::init(Parameter &param)
{
    _feat_dim  = param._feat_dim;
    _class_dim = param._class_dim;
    _epoch     = param._epoch;
    //
    _sample_num = load_data_list(param._input_list, param._input_type);
    _data_mat->toGpu();
    _cent_mat = new cuMatrix<float>(_class_dim, _feat_dim, 1);
    // choose initial centroids
    memcpy(_cent_mat->getHost(), _data_mat->getHost(), sizeof(float) * _class_dim * _feat_dim);
    _cent_mat->toGpu();

    _assigned_vec = new cuMatrix<int>(_sample_num, 1, 1);
    _min_distance = new cuMatrix<float>(_sample_num, 1, 1);
    _distance = new cuMatrix<float>(_sample_num, _class_dim, 1);
    _ave_distance = new cuMatrix<float>(1, 1, 1);

    float ave_dist = assign_centroid();
    INFO("init average distance = %f\n", ave_dist);
}

int cukmeans::load_data_list(char *file_name, char *type)
{
    FILE *fp = fopen(file_name, "rt");
    if (fp == NULL) {
        ERROR("Cannot open file %s to read!\n", file_name);
        return RET_ERROR;
    }
    int sample_num = 0;
    char content[1024];
    while (!feof(fp)) {
        memset(content, '\0', 1024);
        fgets(content, 1024, fp);
        char *temp = strtok(content, " \r\n");
        if (temp == NULL) {
            continue;
        }
        if (strcmp(type, "txt") == 0) {
            sample_num += load_txt_data(temp);
        }
        else {
            sample_num += load_bin_data(temp);
        }
        //
        _name_vec.push_back(temp);
        _sample_vec.push_back(sample_num);
    }
    fclose(fp);
    return sample_num;
}

int cukmeans::load_txt_data(char *file_name)
{
    FILE *fp = fopen(file_name, "rt");
    if (fp == NULL) {
        ERROR("Cannot open file %s to read!\n", file_name);
        return RET_ERROR;
    }
    vector<float> data_vec;
    vector<vector<float> > data_mat;

    data_vec.resize(_feat_dim);
    int sample_num = 0;
    char content[10240];
    while (!feof(fp)) {
        memset(content, '\0', 10240);
        fgets(content, 10240, fp);
        char *temp = strtok(content, " \t\r\n");
        if (temp == NULL) {
            continue;
        }
        int idx = 0;
        while (temp) {
            data_vec[idx++] = atof(temp);
            temp = strtok(NULL, " \t\r\n");
            if (idx == _feat_dim && temp != NULL) {
                ERROR("too many features!\n");
                return RET_OK;
            }
        }
        if (idx != _feat_dim) {
            ERROR("idx = %d vs feat_dim = %d\n", idx, _feat_dim);
            return RET_OK;
        }
        data_mat.push_back(data_vec);
        sample_num++;
    }
    fclose(fp);

    //copy to _data_mat
    _data_mat = new cuMatrix<float>(data_mat.size(), _feat_dim, 1);
    for(size_t i = 0; i < data_mat.size(); i++){
        for(size_t j = 0; j < data_mat[i].size(); j++){
            _data_mat->getHost()[i * _feat_dim + j] = data_mat[i][j];
        }
    }

    INFO("Load samples successfully! sample_num = %d\n", sample_num);
    return sample_num;
}

int cukmeans::load_bin_data(char *file_name)
{
    FILE *fp = fopen(file_name, "rb");
    if (fp == NULL) {
        ERROR("Cannot open file %s to read!\n", file_name);
        return RET_ERROR;
    }
    int feat_dim = 0;
    int sample_num = 0;
    fread(&feat_dim, sizeof(int), 1, fp);
    fread(&sample_num, sizeof(int), 1, fp);

    if (feat_dim != _feat_dim) {
        ERROR("wrong feat dim %d vs %d\n", feat_dim, _feat_dim);
    }
    _data_mat = new cuMatrix<float>(sample_num, feat_dim, 1);
    for (int ii = 0; ii < sample_num; ii++) {
        fread(_data_mat->getHost() + ii * feat_dim, sizeof(float), _feat_dim, fp);
    }
    fclose(fp);

    _sample_num = _data_mat->rows;
    //
    INFO("Load samples successfully! sample_num = %d\n", _sample_num);
    //
    return RET_OK;
}

int cukmeans::do_cluster()
{
    for (int ii = 0; ii < _epoch; ii++) {
        calculate_new_centroid();
        float ave_dist = assign_centroid();
        INFO("epoch%d average distance = %f\n", ii, ave_dist);
    }
    return RET_OK;
}

float cukmeans::calculate_distance(
        vector<float> &data_vec, vector<float> &cent_vec)
{
    float distance = 0.0f;
    for (int ii = 0; ii < _feat_dim; ii++) {
        distance += (data_vec[ii] - cent_vec[ii]) * (data_vec[ii] - cent_vec[ii]);
    }
    return distance;
}

/*
* assigned_vec 每个点指向每个中心
*/
__global__ void g_distance(float* data_mat, float* cent_mat, float* distance,
        int sample_num, int class_dim, int feat_dim){
    //  extern __shared__ float _sum[];
    int sample_id = blockIdx.x;
    int class_id = threadIdx.x;
    float* cur_cent_mat = cent_mat + class_id * feat_dim;
    float* cur_data_mat = data_mat + sample_id * feat_dim;
    float dis = 0;
    for(int i = 0; i < feat_dim; i++){
        float cent = cur_cent_mat[i];
        float data = cur_data_mat[i];
        dis += (cent - data) * (cent - data);
    }
    distance[sample_id * class_dim + class_id] = dis;
}

/*
*根据距离计算出每个点离那个中心点最近，并且将距离记录在min_distance中
*注意　class_dim 不要大于1024
*/
__global__ void g_assign_centroid(
        float* distance, 
        float* min_distance,
        int* assign_centroid,
        int sample_num,
        int class_dim){
    extern __shared__ float block_min[];
    int* block_assign_centroid = (int*)block_min + class_dim;

    int class_id = threadIdx.x;
    int sample_id = blockIdx.x;
    block_min[class_id] = distance[sample_id * class_dim + class_id];
    block_assign_centroid[class_id] = class_id;
    //reduce 
    int len = blockDim.x;
    int tid = threadIdx.x;
    while(len != 1)
    {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(tid < (len >> 1))
        {
            float value1 = block_min[tid];
            float value2 = block_min[tid + skip];
            //TODO 是大于等于还是大于？
            if(value1 > value2){
                block_min[tid] = value2;
                block_assign_centroid[tid] = block_assign_centroid[tid + skip];
            }
        }
        else{
            return;
        }
        len = (len + 1) >> 1;
//        if(blockIdx.x == 4){
//            printf("tix %d assign %d dis %f len %d\n", class_id, block_assign_centroid[class_id], 
//                    block_min[class_id], len);
//        }
    }
    if(tid == 0)
    {
        min_distance[sample_id] = block_min[0];
        assign_centroid[sample_id] = block_assign_centroid[0];
    }
}

/*
   reduce sum 操作
 */
__global__ void g_avr_distance(
        float* min_distance,
        float* ave_distance,
        int sample_num){
    extern __shared__ float sum[];
    sum[threadIdx.x] = 0;
    //先累加到sum中
    for(int i = 0; i < sample_num; i += blockDim.x){
        int idx = i + threadIdx.x;
        if(idx < sample_num){
            sum[threadIdx.x] += min_distance[idx];
        }
    }
    //reduce
    __syncthreads();
    int len = blockDim.x;
    while(len != 1)
    {
        __syncthreads();
        int skip = (len + 1) >> 1;
        if(threadIdx.x < (len >> 1))
        {
            sum[threadIdx.x] += sum[threadIdx.x + skip];
        }
        else{
            return;
        }
        len = (len + 1) >> 1;
    }
    if(threadIdx.x == 0)
    {
        ave_distance[0] = sum[0];
    }
}

/*
 * 将每个节点指向距离他最近的中心节点
 * 复杂度O(n^3)
 * 1)计算优化，先计算出每个点到中心的计算
 */
float cukmeans::assign_centroid()
{
    dim3 block_dim(_sample_num);
    dim3 thread_dim(_class_dim);
    if(_class_dim >= 1024){
        printf("error class_dim > 1024");
        exit(0);
    }
    /*
     * TODO 通过shared_memery进行访存优化
     */
    g_distance<<<block_dim, thread_dim, _class_dim * sizeof(float),0>>>(
            _data_mat->getDev(),
            _cent_mat->getDev(),
            _distance->getDev(),
            _sample_num, 
            _class_dim, 
            _feat_dim);
    checkCudaErrors(hipStreamSynchronize(0));
    getLastCudaError("g_distance");
    //    _distance->toCpu();
    //    _distance->print();
    //    exit(0);

    block_dim = dim3(_sample_num);
    thread_dim = dim3(_class_dim);

    g_assign_centroid<<<block_dim, thread_dim, (2 * sizeof(float)) * _class_dim,0>>>(
            _distance->getDev(),
            _min_distance->getDev(),
            _assigned_vec->getDev(),
            _sample_num,
            _class_dim);
    checkCudaErrors(hipStreamSynchronize(0));
    getLastCudaError("g_assign_centroid");
    //_assigned_vec->toCpu();
    //_assigned_vec->print();
    //_min_distance->toCpu();
   // _min_distance->print();
    //exit(0);

    g_avr_distance<<<dim3(1), dim3(256), sizeof(float) * 256,0>>>(
            _min_distance->getDev(),
            _ave_distance->getDev(),
            _sample_num);
    checkCudaErrors(hipStreamSynchronize(0));
    getLastCudaError("g_avr_distance");
    _ave_distance->toCpu();

    return _ave_distance->getHost()[0] / (float)_sample_num;
}

/*
 * 根据每个节点的指向，算出距离的平均值作为新的中心
 */
__global__ void g_calculate_new_centroid(float* data_mat, float* cent_mat, int* assign_centroid,
        int sample_num, int class_dim, int feat_dim){
    int class_id = blockIdx.x;
    int feat_id = threadIdx.x;
    int count = 0;
    extern __shared__ float sum[];
    sum[feat_id] = 0;
    for(int i = 0; i < sample_num; i++){
        if(class_id == assign_centroid[i]){
            sum[feat_id] += data_mat[i * feat_dim + feat_id];
            count += 1;
        }
    }
    if(count == 0 && feat_id == 0){
        printf("error count == 0\n");
    }
    cent_mat[class_id * feat_dim + feat_id] = sum[feat_id] / count;
}

int cukmeans::calculate_new_centroid()
{
    dim3 block_dim = dim3(_class_dim);
    dim3 thread_dim = dim3(_feat_dim);
    //TODO
    if(_feat_dim >= 1024){
        printf("feat_dim > 1024");
        exit(0);
    }
    //_cent_mat->toCpu();
    //_cent_mat->print();
    //printf("\n");
    g_calculate_new_centroid<<<block_dim, thread_dim, sizeof(float) * _feat_dim, 0>>>(
            _data_mat->getDev(),
            _cent_mat->getDev(),
            _assigned_vec->getDev(),
            _sample_num,
            _class_dim,
            _feat_dim);
    checkCudaErrors(hipStreamSynchronize(0));
    getLastCudaError("g_calculate_new_centroid");
//    _cent_mat->toCpu();
//    _cent_mat->print();
//    exit(0);
}

int cukmeans::write()
{
    char out_file_name[1024];
    _assigned_vec->toCpu();
    for (int ii = 0; ii < _sample_vec.size(); ii++) {
        sprintf(out_file_name, "%s.class", _name_vec[ii].c_str());
        FILE *fp = fopen(out_file_name, "wt");
        if (fp == NULL) {
            ERROR("Cannot open file %s to write!\n", out_file_name);
            return RET_ERROR;
        }
        int st = (ii == 0) ? 0 : _sample_vec[ii - 1];
        int ed = _sample_vec[ii];
        for (int jj = st; jj < ed; jj++) {
            fprintf(fp, "%d\n", _assigned_vec->getHost()[jj]);
        }
        fclose(fp);
    }
    return RET_OK;
}
